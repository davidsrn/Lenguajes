#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N (4096*4096)
#define HILOS_POR_BLOQUE 512

__global__ void sumaenlagpu(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < n){
		c[index] = a[index] + b[index];
	}
}

void sumaenlacpu(int *a, int *b, int *c, int n){
	for(int i = 0; i< n; i++){
		c[i] = a[i] + b[i];
	}
}

void numerosAleatorios(int *a, int n){
	for (int i = 0; i < n; i++){
		a[i] = rand() % 100000;
	}
}

int comparaeneteros(int *a, int *b, int n){
	int pass = 0;
	for (int i = 0; i < n; i++){
		if (a[i] != b[i])
			printf("Los valores  en a[%i] = %i y en b[%i] = %i", i, a[i], i, b[i]);
	}
	if (pass == 0){
		printf("Comprobacion aceptada \n");
	}
	else{
		printf("No paso la comprobacion \n");
	}
	return pass;
}

int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int tam = N*sizeof(int);
	//reserva memoria en DEVICE gpu
	hipMalloc((void**)&d_a, tam);
	hipMalloc((void**)&d_b, tam);
	hipMalloc((void**)&d_c, tam);

	// reserva de memoria en HOST cpu
	a = (int*)malloc(tam);
	b = (int*)malloc(tam);
	c = (int*)malloc(tam);

	//inicializamos con aleatorios
	numerosAleatorios(a,N);
	numerosAleatorios(b,N);

	//copiamos valores de cpu 'a' a gpu 'd_a'
	hipMemcpy(d_a, a, tam, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tam, hipMemcpyHostToDevice);

	//ejecutamos la funcion clock_para tomar una muestra del tiempo
	clock_t tiempogpu = clock();

	//ejecucion del kernel

	sumaenlagpu << <N / HILOS_POR_BLOQUE, HILOS_POR_BLOQUE >> >(d_a, d_b, d_c, N);
	printf("Tiempo transcurrido al procesador en GPU: %f\n", ((double)clock() - tiempogpu) / CLOCKS_PER_SEC);
	//copio resultado de la gpu al cpu
	hipMemcpy(c, d_c, tam, hipMemcpyDeviceToHost);

	int *c_h;
	c_h = (int *)malloc(tam);

	clock_t tiempocpu = clock();
	sumaenlacpu(a, b, c_h, N);
	printf("Tiempo transcurrido al procesador en CPU: %f\n", ((double)clock() - tiempocpu) / CLOCKS_PER_SEC);
	comparaeneteros(c, c_h, N);

	//borramos
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
