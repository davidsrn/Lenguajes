// David Ramirez A01206423
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define RECTS 1e9
#define BLOCKS 1000
#define THREADS 512

// long num_rects = 100000, i;
// double mid, height, width, area;
// sum = 0.0;
// width = 1.0 / (double) num_rects;
// for (i = 0; i < num_rects; i++){
//   mid = (i + 0.5) * width;
//   height = 4.0 / (1.0 + mid * mid);
//   sum += height;
// }
// area = width * sum;


//This function is for the GPU:
__global__ void piCalc(double *area, double width, int rects) {
	double mid, height;
	// Get our index
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	// Pos in array
	int id = index;
	// do while we are inside our array
	while(index<rects){
		//Original pi algo
		mid = (index + 0.5) * width;
    height = 4.0 / (1.0 + mid * mid);
		area[id] += height;
		// Move our index
		index += (blockDim.x*gridDim.x);
	}
}

int main(){
	// Normal Array
	double *pi;
	// GPU Array
	double *d_pi;
	// Dimention of our threads
	int size=(BLOCKS*THREADS);
	// Result var
	double area=0;
	// with var initialization
	double width=1.0/(double) RECTS;

	pi = (double*) malloc(size*sizeof(double));

	hipMalloc((void **)&d_pi, size * sizeof(double));//Device memory (GPU)
	// Send vars to GPU
  hipMemcpy(d_pi, pi, size * sizeof(double), hipMemcpyHostToDevice);
	// Do the operation in the GPU
	piCalc<<<BLOCKS, THREADS>>>(d_pi, width, RECTS); // Launch GPU with its corresponding inputs
	// Retrieve the results
	hipMemcpy(pi, d_pi, size * sizeof(double), hipMemcpyDeviceToHost); // Copy output array from GPU back to CPU (Device to host)
	// Sum all of our values
	for(int i = 0; i<size; ++i){
    area += pi[i];
  }
	// Calc the area
	area=width*area;

	printf("Pi = %lf\n", area);
	// Free our CPUs and GPUs
	free(pi);
	hipFree(d_pi);

	return 0;
}
