
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10
#define M 10
#define ThreadsPerBlock 16
#define NumBlocks  (ThreadsPerBlock + (N*M-1))/ThreadsPerBlock

__device__ void convolution(int conv_col, int conv_row, float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
	int conv_index = conv_col+ conv_row*max_col;
	d_conv[conv_index] = 0;
	for(int k_row = 0;  k_row < k_size; k_row ++){
			for(int k_col = 0;  k_col < k_size ; k_col ++){
				d_conv[conv_index] +=  
				d_kernel[k_col + (k_row*k_size)] *
				d_matrix[(conv_col+k_col) + (conv_row+k_row)*size_x];
				//		printf("row %i col %i d_conv[] = %f \n", row, col, d_conv[col+ row*max_col]);
			}
		}
}

__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	
	if(max_row > row && max_col > col){
		convolution(col, row, d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
	}
}

void print_mat(float *mat, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			printf("%.1f\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}


void fill_mat(float *mat, int n){
	int c = 0;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			mat[i*n+j] = c++;
		}
	}
}

int main(){
	float *h_kernel, *h_matrix, *h_conv;
	float *d_kernel, *d_matrix, *d_conv;

	int k_size = 5;
	int size_x = N;
	int size_y = M;
	int max_row = size_x - (k_size/2)*2;
	int max_col = size_y - (k_size/2)*2;

	h_kernel = (float *)malloc(sizeof(float)*k_size*k_size);
	h_matrix = (float *)malloc(sizeof(float)*size_x*size_y);
	h_conv = (float *)malloc(sizeof(float)*max_row*max_col);

	fill_mat(h_kernel, k_size);
	fill_mat(h_matrix, size_x);

	print_mat(h_kernel, k_size);
	print_mat(h_matrix, size_x);

	
	hipMalloc((void**)&d_kernel,sizeof(float)*k_size*k_size);
	hipMalloc((void**)&d_matrix,sizeof(float)*size_x*size_y);
	hipMalloc((void**)&d_conv,sizeof(float)*max_row*max_col);

	hipMemcpy(d_kernel, h_kernel,sizeof(float)*k_size*k_size, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix, h_matrix,sizeof(float)*size_x*size_y, hipMemcpyHostToDevice);

	dim3 Blocks(NumBlocks,NumBlocks);
	dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);
	
	//printf("Blocks %i \nThreads %i \n", NumBlocks, ThreadsPerBlock);
	valid_convolution<<<Blocks, Threads>>>(d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
 
	hipMemcpy(h_conv, d_conv,sizeof(float)*max_row*max_col, hipMemcpyDeviceToHost);
	
	print_mat(h_conv, max_col);
	
	free(h_kernel);
	free(h_conv);
	free(h_matrix);
	
	hipFree(d_kernel);
	hipFree(d_conv);
	hipFree(d_matrix);
}