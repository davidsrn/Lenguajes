#include "hip/hip_runtime.h"
/*
 * BasicSum.cu
 *
 *  Created on: 9/23/2019
 *      Author: Benjamin Valdes
 *		ID: A0082900
*/

/*
Simple example of addition using C++ CUDA
*/

#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void sum(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);
	//allocating memory
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	//get values from user
	printf("please give me 2 values\n");
	scanf("%i %i", &a, &b);
	//copy values from cpu(RAM) to the graphics card memmory
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	//run kernel
	sum << <1, 1 >> >(d_a, d_b, d_c);
	//get values from memory card
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("the result is %d\n", c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
