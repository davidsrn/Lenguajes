// David Ramirez A01206423
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define N 10
#define M 10
#define WIDTH 2

// Device mat mult
__global__ void MatrixMul(float *darray_1, float *darray_2 , float *dres_arr, int n){
  // cols and rows definition
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  // Mat mult operation
  for(int i = 0; i<n; i++){
    dres_arr[row*n+col]+= darray_1[row*n+i]*darray_2[col+i*n];
    // printf("row %i * height %i col %i index %i res %f\n", row, n, col, i, dres_arr[row*n+col]);
  }
}

int main(){
  float ThreadsPerBlock = 16;
  float NumBlocks = (ThreadsPerBlock + (N*M-1))/ThreadsPerBlock;
  float array_1[WIDTH][WIDTH] ,array_2[WIDTH][WIDTH], res_arr_m[WIDTH][WIDTH];
  float *darray_1 , *darray_2 ,*dres_arr;
  //Fill arrays
  // for(int i = 0; i<WIDTH ; i++){
  //    for(int j = 0; j<WIDTH ; j++){
  //       array_1[i][j] = 2;
  //       array_2[i][j] = 2;
  //    }
  // }
  printf("original mats\n");

  array_1[0][0] = 1;
  array_1[0][1] = 2;
  array_1[1][0] = 3;
  array_1[1][1] = 4;

  array_2[0][0] = 4;
  array_2[0][1] = 5;
  array_2[1][0] = 6;
  array_2[1][1] = 7;
// print array values
  for(int i = 0; i<WIDTH; i++){
    for(int j = 0; j < WIDTH; j++){
      printf("%f ", array_1[i][j]);
    }
    printf("\n");
  }
  printf("\n");
  for(int i = 0; i<WIDTH; i++){
    for(int j = 0; j < WIDTH; j++){
      printf("%f ", array_2[i][j]);
    }
    printf("\n");
  }
  printf("\n");

// Create device arrays
  hipMalloc((void**) &darray_1, WIDTH*WIDTH*sizeof(int));
  hipMalloc((void**) &darray_2, WIDTH*WIDTH*sizeof(int));

// Send arrays to device
  hipMemcpy(darray_1, array_1, WIDTH*WIDTH*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(darray_2, array_2, WIDTH*WIDTH*sizeof(int), hipMemcpyHostToDevice);

// Save device space for res array
  hipMalloc((void**) &dres_arr, WIDTH*WIDTH*sizeof(int));

// Call kernel
  dim3 Blocks(NumBlocks,NumBlocks);
  dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);

  MatrixMul<<<Blocks,Threads>>>(darray_1, darray_2, dres_arr, WIDTH);

// Save result to host
  hipMemcpy(res_arr_m , dres_arr, WIDTH*WIDTH*sizeof(int), hipMemcpyDeviceToHost);

// Print res
  printf("result\n");
  for(int i = 0; i<WIDTH; i++){
    for(int j = 0; j < WIDTH; j++){
      printf("%f ", res_arr_m[i][j]);
    }
    printf("\n");
  }
}
